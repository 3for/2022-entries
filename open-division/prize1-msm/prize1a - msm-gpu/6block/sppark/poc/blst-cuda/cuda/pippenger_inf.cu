// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377.hpp>
#elif defined(FEATURE_BN254)
# include <ff/alt_bn128.hpp>
#else
# error "no FEATURE"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__
extern "C"
RustError mult_pippenger_inf(point_t* out, const affine_t points[], size_t npoints,
                                           const scalar_t scalars[],
                             size_t ffi_affine_sz)
{
    pippinger_t<bucket_t, point_t, affine_t, scalar_t> pipp;
    return pipp.msm(*out, points, npoints, scalars, false, ffi_affine_sz);
}
#endif
