// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ff/bls12-381.hpp>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__
extern "C"
RustError mult_pippenger(point_t* out, const affine_t points[], size_t npoints,
                                       const scalar_t scalars[])
{
    pippinger_t<bucket_t, point_t, affine_t, scalar_t> pipp;
    return pipp.msm(*out, points, npoints, scalars, false);
}
#endif
